
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

int M,N;
double *A, *AT;
double *d_A, *d_AT;

__global__ void MT(double *A, double *AT, int m, int n){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if(idx < m){
		for(int rows=0; rows<n; rows++){
			AT[idx * n + rows] = A[idx + rows * m];
		}
	}
}

int main(int argc, char *argv[]){

	M = atoi(argv[1]);
	N = atoi(argv[2]);

	A = (double *)malloc(M*N*sizeof(double));
	AT = (double *)malloc(M*N*sizeof(double));

	for(int i=0;i<M*N;i++) A[i] = i;
	for(int i=0;i<M*N;i++) AT[i] = 0;

	hipMalloc((void**) &d_A, M*N*sizeof(double));
	hipMalloc((void**) &d_AT, M*N*sizeof(double));

	hipMemcpy(d_A, A, M*N*sizeof(double), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	dim3 block(256);
	dim3 grid((255+M)/256);

	hipEventRecord(start);
	for(int i=0;i<5;i++) MT<<<grid,block>>>(d_A,d_AT,M,N);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // this is the runtime for 100 spmvs in ms
    printf("runtime [ms]: %f\n", milliseconds/ 5.0 );

	hipMemcpy(AT, d_AT, M*N*sizeof(double), hipMemcpyDeviceToHost);

	free(A);
	free(AT);
	hipFree(d_A);
	hipFree(d_AT);

}
